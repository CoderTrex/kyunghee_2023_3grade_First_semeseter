#include "hip/hip_runtime.h"
#include "" 
#include <hip/hip_runtime.h> 
#include <stdlib.h> 
#include <stdio.h> 
#define SIZE (2048 * 2048)
#define THREADS_PER_BLOCK 512

// // __global__을 통해서 커널임을 표시한다. host에서 호출된다. 
// __global__ void dot(int *a, int *b, int *c, int n){ 
//     // 수많은 블록과 스레드가 동시에 처리한다. 
//     // 위에서 정의한 index를 통해서 스레드들을 구별한다. 
//     int index = threadIdx.x + blockIdx.x * blockDim.x;
// 	c[index] = a[index] * b[index];
//     printf("%d = %d * %d\n", c[index], a[index], b[index]); 
// } 

__global__ void dot(int *a, int *b, int *c)
{
    __shared__ int      temp[THREADS_PER_BLOCK];
    int                 index = threadIdx.x + blockIdx.x * blockDim.x;
    
    temp[threadIdx.x] = a[index] * b[index];
    
    __syncthreads();

    if(threadIdx.x == 0){
        int sum = 0;
        for(int i = 0 ; i < THREADS_PER_BLOCK ; i++){
            sum += temp[i];
        }
        atomicAdd(c, sum);
    }
}

int main(){ 
    int *a, *b, *c; 
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);
    
    // 호스트의 메모리에 할당한다. 
    a = (int *)malloc(size);
    b = (int *)malloc(size); 
    c = (int *)malloc(size); 
    
    // hipMalloc(destination, number of byte)로 device의 메모리를 할당한다.
    hipMalloc(&d_a, size); 
    hipMalloc(&d_b, size); 
    hipMalloc(&d_c, size); 
    
    // 초기화 
    for (int i = 0; i<SIZE; ++i) { 
        a[i] = i; 
        b[i] = i; 
        c[i] = 0; 
    } 
    // hipMemcpy(destination, source, number of byte, hipMemcpyHostToDevice)로 호스트에서 디바이스로 메모리를 카피한다. 
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size, hipMemcpyHostToDevice); 
    
    // 함수 호출을 위해서 새로운 신텍스 요소를 추가할 필요가 있다. 
    // 첫번째 parameter는 블럭의 수이다. 예제에서는 스레드 블럭이 하나이다. 
    // SIZE는 1024개의 스레드를 의미한다. 
    dot <<< SIZE / THREADS_PER_BLOCK , THREADS_PER_BLOCK >>>(d_a, d_b, d_c, SIZE); 
    
    //hipMemcpy(source, destination, number of byte, cudaMemDeviceToHost)로 디바이스의 메모리(연산 결과 데이터)를 호스트에 카피한다. 
    hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost); 
    hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost); 
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost); 
    
    for (int i = 0; i<SIZE; ++i) {
        printf("c[%d] = %d\n", i, c[i]);
	}
    
    // 호스트의 메모리 할당 해제 
    free(a); 
    free(b); 
    free(c); 
    // hipFree(d_a)를 통해 디바이스의 메모리를 할당 해제 
    hipFree(d_a); 
    hipFree(d_b); 
    hipFree(d_c); 
    
    return 0;
}